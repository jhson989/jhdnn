#include <jhdnn.cuh>
#include <vector>
#include <algorithm>

int main(void) {


    // input
    float* d_input;
    std::vector<float> input(3*1*128*128, 0);
    std::generate( input.begin(), input.end(), [](){ return ((std::rand()%101-50)%10); } );
    cudaErrChk( hipMalloc(&d_input, sizeof(float)*3*1*128*128) );
    cudaErrChk( hipMemcpy(d_input, input.data(), sizeof(float)*3*1*128*128, hipMemcpyHostToDevice) );
    
    cudnn_create();
    cuConvFloat conv_cu(
        3, 3, 128, 128,
        3, 128, 128,
        3, 3
    );
    jhConvFloat conv_jh(
        3, 3, 128, 128,
        3, 128, 128,
        3, 3
    );

    // filter
    float filter[] = {
        -1, -1, -1, -1, 9, -1, -1, -1, -1,
        0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0,
        -1, -1, -1, -1, 9, -1, -1, -1, -1,
        0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0,
        -1, -1, -1, -1, 9, -1, -1, -1, -1
    };
    conv_cu.set_weights(filter);
    conv_jh.set_weights(filter);

    float* d_dy;
    std::vector<float> dy(3*1*128*128, 1);
    cudaErrChk( hipMalloc(&d_dy, sizeof(float)*3*1*128*128) );
    cudaErrChk( hipMemcpy(d_dy, dy.data(), sizeof(float)*3*1*128*128, hipMemcpyHostToDevice) );
    cudaErrChk( hipDeviceSynchronize() );

    conv_cu.forward(d_input);
    conv_cu.backward(d_dy);


    conv_jh.forward(d_input);
    conv_jh.backward(d_dy);

    cudnn_destroy();
    return 0;       
}