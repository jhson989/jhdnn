

#include <jhdnn.cuh>
#include <vector>
#include <algorithm>

int main(void) {
    
    cudnn_create();
    cuConvFloat conv(
        128, 3, 128, 128,
        3, 128, 128,
        3, 3
    );
    
    float filter[] = {
        -1, -1, -1, -1, 9, -1, -1, -1, -1,
        0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0,
        -1, -1, -1, -1, 9, -1, -1, -1, -1,
        0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0,
        -1, -1, -1, -1, 9, -1, -1, -1, -1
    };

    float* d_input;
    std::vector<float> input(3*1*128*128, 0);
    std::generate( input.begin(), input.end(), [](){ return ((std::rand()%101-50)%10); } );
    cudaErrChk( hipMalloc(&d_input, sizeof(float)*3*1*128*128) );
    cudaErrChk( hipMemcpy(d_input, input.data(), sizeof(float)*3*1*128*128, hipMemcpyHostToDevice) );
    conv.set_weights(filter);

    float* d_dy;
    std::vector<float> dy(3*1*128*128, 1);
    cudaErrChk( hipMalloc(&d_dy, sizeof(float)*3*1*128*128) );
    cudaErrChk( hipMemcpy(d_dy, dy.data(), sizeof(float)*3*1*128*128, hipMemcpyHostToDevice) );

    conv.forward(d_input);
    conv.backward(d_dy);

    cudnn_destroy();
    return 0;       
}