#include <jhdnn.cuh>

/***************************************************************
 * Debug code
 ***************************************************************/
void cudnnAssert(hipdnnStatus_t code, const char *file, int line) {
   if (code != HIPDNN_STATUS_SUCCESS) {
        fprintf(stderr,"cuDNN assert: %s %s %d\n", hipdnnGetErrorString(code), file, line);
   }
}

void cudaAssert(hipError_t code, const char *file, int line) {
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
   }
}




/***************************************************************
 * Layer
 ***************************************************************/
cuLayerFloat::cuLayerFloat() {


}


cuLayerFloat::~cuLayerFloat() {


}

void cuLayerFloat::forward(float* input) {

}

void cuLayerFloat::backward(float* input_grad) {

}