#include <jhdnn.cuh>


hipdnnHandle_t* cudnn;

/***************************************************************
 * Debug code
 ***************************************************************/
void cudnnAssert(hipdnnStatus_t code, const char *file, int line) {
    if (code != HIPDNN_STATUS_SUCCESS) {
        fprintf(stderr,"cuDNN assert: %s %s %d\n", hipdnnGetErrorString(code), file, line);
        exit(1);
    }
}

void cudaAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(1);
    }
}

void cudnn_create() {

    /******************************************************************
     * 1. Define cudnn Handler
     *******************************************************************/
    cudnn = new hipdnnHandle_t;
    cudnnErrChk( hipdnnCreate(cudnn) );
};

void cudnn_destroy() {
    cudnnErrChk( hipdnnDestroy(*cudnn) );
    delete cudnn ;
}
