#include <jhdnn.cuh>

extern hipdnnHandle_t* cudnn;


cuConvFloat::cuConvFloat(

    const int BATCH_NUM_, 
    const int INPUT_C_, const int INPUT_H_,const int INPUT_W_, 
    const int OUTPUT_C_, const int OUTPUT_H_, const int OUTPUT_W_,
    const int FILTER_H_, const int FILTER_W_, 
    const int PAD_H_, const int PAD_W_, 
    const int STRIDE_H_, const int STRIDE_W_,
    const int DILATION_H_, const int DILATION_W_
    
) : BATCH_NUM(BATCH_NUM_), INPUT_C(INPUT_C_), INPUT_H(INPUT_H_), INPUT_W(INPUT_W_), FILTER_H(FILTER_H_), FILTER_W(FILTER_W_), PAD_H(PAD_H_), PAD_W(PAD_W_), STRIDE_H(STRIDE_H_), STRIDE_W(STRIDE_W_), OUTPUT_C(OUTPUT_C_), OUTPUT_H(OUTPUT_H_), OUTPUT_W(OUTPUT_W_), DILATION_H(DILATION_H_), DILATION_W(DILATION_W_)
{

    /******************************************************************
     * 1. Define cudnn Handler
     *******************************************************************/
    if (cudnn == nullptr) {
        cudnn = new hipdnnHandle_t;
        cudnnErrChk( hipdnnCreate(cudnn) );
    }


    /******************************************************************
     * 2. Describe Conv2D operands
     *    - Input tensor : size, layout
     *    - Output tensor : size, layout
     *******************************************************************/
    cudnnErrChk( hipdnnCreateTensorDescriptor(&desc_input) );
    cudnnErrChk( hipdnnSetTensor4dDescriptor(
        desc_input,
        /*LAYOUT*/HIPDNN_TENSOR_NCHW, /*DATATYPE*/data_type, /*N*/BATCH_NUM, /*C*/ INPUT_C, /*H*/INPUT_H, /*W*/INPUT_W
    ) );

    cudnnErrChk( hipdnnCreateTensorDescriptor(&desc_output) );
    cudnnErrChk( hipdnnSetTensor4dDescriptor(
        desc_output,
        /*LAYOUT*/HIPDNN_TENSOR_NCHW, /*DATATYPE*/data_type, /*N*/BATCH_NUM, /*C*/ OUTPUT_C, /*H*/OUTPUT_H, /*W*/OUTPUT_W
    ) );




    /******************************************************************
     * 3. Describe Conv2D kernel
     *    - Filter : layout, size
     *    - Conv2D layer : pad, stride, dilation, etc.
     *    - Conv2d algorithm
     *******************************************************************/
    // Filter (weights)
    cudnnErrChk( hipdnnCreateFilterDescriptor(&desc_filter) );
    cudnnErrChk( hipdnnSetFilter4dDescriptor(
        desc_filter,
        /*DATATYPE*/data_type, /*LAYOUT*/HIPDNN_TENSOR_NCHW, /*OUT_CH*/OUTPUT_C, /*IN_CH*/ INPUT_C, /*KERNEL_H*/FILTER_H, /*KERNEL_W*/FILTER_W
    ) );

    // Layer 
    cudnnErrChk( hipdnnCreateConvolutionDescriptor(&desc_conv2d) );
    cudnnErrChk( hipdnnSetConvolution2dDescriptor(
        desc_conv2d,
        /*PAD_H*/PAD_H, /*PAD_W*/PAD_W, /*STRIDE_VERTICAL*/STRIDE_H, /*STRIDE_HORIZONTAL*/STRIDE_W, /*DILATION_H*/DILATION_H, /*DILATION_W*/DILATION_W, /*MODE*/HIPDNN_CROSS_CORRELATION, /*DATATYPE*/data_type
    ) );

    // Algorithm
    cudnnErrChk( hipdnnFindConvolutionForwardAlgorithm(
        *cudnn, desc_input, desc_filter, desc_conv2d, desc_output, 1, &num_conv2d_algo, &perf_conv2d_algo
    ) );


    /******************************************************************
     * 4. Calculate work-space size
     *******************************************************************/
    cudnnErrChk( hipdnnGetConvolutionForwardWorkspaceSize(*cudnn, desc_input, desc_filter, desc_conv2d, desc_output, perf_conv2d_algo.algo, &bytes_workspace) );
  

    /******************************************************************
     * 5. Allocate memory
     *    - work-space
     *    - HOST : input, output, kernel
     *    - GPU : input, output, kernel
     *******************************************************************/
    cudaErrChk (hipMalloc (&d_workspace, bytes_workspace));
 
    h_input = (float*) malloc(sizeof(float)*BATCH_NUM*INPUT_C*INPUT_H*INPUT_W);
    h_output = (float*) malloc(sizeof(float)*BATCH_NUM*OUTPUT_C*OUTPUT_H*OUTPUT_W);
    h_filter = (float*) malloc(sizeof(float)*OUTPUT_C*INPUT_C*FILTER_H*FILTER_W);
    cudaErrChk( hipMalloc(&d_input, sizeof(float)*BATCH_NUM*INPUT_C*INPUT_H*INPUT_W) );
    cudaErrChk( hipMalloc(&d_output, sizeof(float)*BATCH_NUM*OUTPUT_C*OUTPUT_H*OUTPUT_W) );
    cudaErrChk( hipMalloc(&d_filter, sizeof(float)*OUTPUT_C*INPUT_C*FILTER_H*FILTER_W) );

    /******************************************************************
     * 6. Initialize filter
     *******************************************************************/
    std::generate(h_filter, h_filter+OUTPUT_C*INPUT_C*FILTER_H*FILTER_W, [](){return (std::rand()%101-50)/10;});
    cudaErrChk( hipMemcpy(d_filter, h_filter, sizeof(float)*OUTPUT_C*INPUT_C*FILTER_H*FILTER_W, hipMemcpyHostToDevice) );
}




cuConvFloat::~cuConvFloat() {

    /******************************************************************
     * Finallize
     *******************************************************************/
     free (h_input);
     free (h_output);
     free (h_filter);
 
     cudaErrChk( hipFree(d_workspace) );
     cudaErrChk( hipFree(d_input) );
     cudaErrChk( hipFree(d_output) );
     cudaErrChk( hipFree(d_filter) );
 
     cudnnErrChk( hipdnnDestroyTensorDescriptor(desc_input) );
     cudnnErrChk( hipdnnDestroyTensorDescriptor(desc_output) );
     cudnnErrChk( hipdnnDestroyFilterDescriptor(desc_filter) );

}


void cuConvFloat::forward(float* input) {

    /******************************************************************
     * 6. Launch kernel
     *******************************************************************/
    const float alpha=1, beta=0;
    cudnnErrChk( hipdnnConvolutionForward(*cudnn
                                        , /*ALPHA*/&alpha
                                        , /*INPUT*/desc_input, d_input
                                        , /*KERNEL*/desc_filter, d_filter
                                        , /*LAYER*/desc_conv2d, perf_conv2d_algo.algo, d_workspace, bytes_workspace
                                        , /*BETA*/&beta
                                        , /*OUTPUT*/desc_output, d_output
                                    ) );
 


    /******************************************************************
     * 7. Get result
     *******************************************************************/
    cudaErrChk( hipMemcpy(h_output, d_output, sizeof(float)*BATCH_NUM*OUTPUT_C*OUTPUT_H*OUTPUT_W, hipMemcpyDeviceToHost) );

}

void cuConvFloat::backward(float* back_grad) {

}

void cuConvFloat::set_weight(float* filter_) {
    memcpy(h_filter, filter_, sizeof(float)*OUTPUT_C*INPUT_C*FILTER_H*FILTER_W);
    cudaErrChk( hipMemcpy(d_filter, h_filter, sizeof(float)*OUTPUT_C*INPUT_C*FILTER_H*FILTER_W, hipMemcpyHostToDevice) );
}