#include <jhdnn.cuh>

extern hipdnnHandle_t* cudnn;


cuConvFloat::cuConvFloat(

    const int BATCH_NUM_, 
    const int INPUT_C_, const int INPUT_H_, const int INPUT_W_, 
    const int OUTPUT_C_, const int FILTER_H_, const int FILTER_W_, 
    const int PAD_H_, const int PAD_W_, 
    const int STRIDE_H_, const int STRIDE_W_,
    const int DILATION_H_, const int DILATION_W_
    
) : BATCH_NUM(BATCH_NUM_), INPUT_C(INPUT_C_), INPUT_H(INPUT_H_), INPUT_W(INPUT_W_), OUTPUT_C(OUTPUT_C_), FILTER_H(FILTER_H_), FILTER_W(FILTER_W_), PAD_H(PAD_H_), PAD_W(PAD_W_), STRIDE_H(STRIDE_H_), STRIDE_W(STRIDE_W_), DILATION_H(DILATION_H_), DILATION_W(DILATION_W_)
{
    OUTPUT_H=(INPUT_H-FILTER_H+2*PAD_H)/STRIDE_H + 1;
    OUTPUT_W=(INPUT_W-FILTER_W+2*PAD_W)/STRIDE_W + 1;
    /******************************************************************
     * 2. Describe Conv2D operands
     *    - Input tensor : size, layout
     *    - Output tensor : size, layout
     *    - dx tensor : size, layout
     *    - dy tensor : size, layout
     *******************************************************************/
    cudnnErrChk( hipdnnCreateTensorDescriptor(&desc_x) );
    cudnnErrChk( hipdnnSetTensor4dDescriptor(
        desc_x,
        /*LAYOUT*/HIPDNN_TENSOR_NCHW, /*DATATYPE*/data_type, /*N*/BATCH_NUM, /*C*/ INPUT_C, /*H*/INPUT_H, /*W*/INPUT_W
    ) );

    cudnnErrChk( hipdnnCreateTensorDescriptor(&desc_y) );
    cudnnErrChk( hipdnnSetTensor4dDescriptor(
        desc_y,
        /*LAYOUT*/HIPDNN_TENSOR_NCHW, /*DATATYPE*/data_type, /*N*/BATCH_NUM, /*C*/ OUTPUT_C, /*H*/OUTPUT_H, /*W*/OUTPUT_W
    ) );

    cudnnErrChk( hipdnnCreateTensorDescriptor(&desc_dx) );
    cudnnErrChk( hipdnnSetTensor4dDescriptor(
        desc_dx,
        /*LAYOUT*/HIPDNN_TENSOR_NCHW, /*DATATYPE*/data_type, /*N*/BATCH_NUM, /*C*/ INPUT_C, /*H*/INPUT_H, /*W*/INPUT_W
    ) );

    cudnnErrChk( hipdnnCreateTensorDescriptor(&desc_dy) );
    cudnnErrChk( hipdnnSetTensor4dDescriptor(
        desc_dy,
        /*LAYOUT*/HIPDNN_TENSOR_NCHW, /*DATATYPE*/data_type, /*N*/BATCH_NUM, /*C*/ OUTPUT_C, /*H*/OUTPUT_H, /*W*/OUTPUT_W
    ) );


    /******************************************************************
     * 3. Describe Conv2D kernel
     *    - Filter : layout, size
     *    - Conv2D layer : pad, stride, dilation, etc.
     *    - Conv2d forward algorithm
     *    - Conv2d backward algorithm
     *******************************************************************/
    // Filter (weights)
    cudnnErrChk( hipdnnCreateFilterDescriptor(&desc_filter) );
    cudnnErrChk( hipdnnSetFilter4dDescriptor(
        desc_filter,
        /*DATATYPE*/data_type, /*LAYOUT*/HIPDNN_TENSOR_NCHW, /*OUT_CH*/OUTPUT_C, /*IN_CH*/ INPUT_C, /*KERNEL_H*/FILTER_H, /*KERNEL_W*/FILTER_W
    ) );

    // Filter (weights) backward
    cudnnErrChk( hipdnnCreateFilterDescriptor(&desc_dw) );
    cudnnErrChk( hipdnnSetFilter4dDescriptor(
        desc_dw,
        /*DATATYPE*/data_type, /*LAYOUT*/HIPDNN_TENSOR_NCHW, /*OUT_CH*/OUTPUT_C, /*IN_CH*/ INPUT_C, /*KERNEL_H*/FILTER_H, /*KERNEL_W*/FILTER_W
    ) );

    // Layer 
    cudnnErrChk( hipdnnCreateConvolutionDescriptor(&desc_conv2d) );
    cudnnErrChk( hipdnnSetConvolution2dDescriptor(
        desc_conv2d,
        /*PAD_H*/PAD_H, /*PAD_W*/PAD_W, /*STRIDE_VERTICAL*/STRIDE_H, /*STRIDE_HORIZONTAL*/STRIDE_W, /*DILATION_H*/DILATION_H, /*DILATION_W*/DILATION_W, /*MODE*/HIPDNN_CROSS_CORRELATION, /*DATATYPE*/data_type
    ) );

    // Forward algorithm
    cudnnErrChk( hipdnnFindConvolutionForwardAlgorithm(
        *cudnn, desc_x, desc_filter, desc_conv2d, desc_y, 1, &num_conv2d_algo_forward, &perf_conv2d_algo_forward
    ) );

    // Backward algorithm
    cudnnErrChk( hipdnnFindConvolutionBackwardDataAlgorithm(
        *cudnn, desc_filter, desc_dy, desc_conv2d, desc_dx, 1, &num_conv2d_algo_backward_data, &perf_conv2d_algo_backward_data
    ) );
    cudnnErrChk( hipdnnFindConvolutionBackwardFilterAlgorithm(
        *cudnn, desc_x, desc_dy, desc_conv2d, desc_dw, 1, &num_conv2d_algo_backward_filter, &perf_conv2d_algo_backward_filter
    ) );

    /******************************************************************
     * 4. Calculate work-space size for forward and backward
     *******************************************************************/
    cudnnErrChk( hipdnnGetConvolutionForwardWorkspaceSize(*cudnn, desc_x, desc_filter, desc_conv2d, desc_y, perf_conv2d_algo_forward.algo, &bytes_workspace_forward) );
    cudnnErrChk( hipdnnGetConvolutionBackwardDataWorkspaceSize(*cudnn, desc_filter, desc_dy, desc_conv2d, desc_dx, perf_conv2d_algo_backward_data.algo, &bytes_workspace_backward_data) );
    cudnnErrChk( hipdnnGetConvolutionBackwardFilterWorkspaceSize(*cudnn, desc_x, desc_dy, desc_conv2d, desc_dw, perf_conv2d_algo_backward_filter.algo, &bytes_workspace_backward_filter) );
    
    /******************************************************************
     * 5. Allocate memory
     *    - work-space
     *    - HOST : x, y, dx, dy, kernel -> Not necessary
     *    - GPU : x, y, dx, dy, kernel
     *******************************************************************/
    cudaErrChk (hipMalloc (&d_workspace_forward, bytes_workspace_forward));
    cudaErrChk (hipMalloc (&d_workspace_backward_data, bytes_workspace_backward_data));
    cudaErrChk (hipMalloc (&d_workspace_backward_filter, bytes_workspace_backward_filter));
 
    h_filter = (float*) malloc(sizeof(float)*OUTPUT_C*INPUT_C*FILTER_H*FILTER_W);
    cudaErrChk( hipMalloc(&d_x, sizeof(float)*BATCH_NUM*INPUT_C*INPUT_H*INPUT_W) );
    cudaErrChk( hipMalloc(&d_y, sizeof(float)*BATCH_NUM*OUTPUT_C*OUTPUT_H*OUTPUT_W) );
    cudaErrChk( hipMalloc(&d_dx, sizeof(float)*BATCH_NUM*INPUT_C*INPUT_H*INPUT_W) );
    cudaErrChk( hipMalloc(&d_dy, sizeof(float)*BATCH_NUM*OUTPUT_C*OUTPUT_H*OUTPUT_W) );
    cudaErrChk( hipMalloc(&d_filter, sizeof(float)*OUTPUT_C*INPUT_C*FILTER_H*FILTER_W) );
    cudaErrChk( hipMalloc(&d_dw, sizeof(float)*OUTPUT_C*INPUT_C*FILTER_H*FILTER_W) );
    /******************************************************************
     * 6. Initialize filter
     *******************************************************************/
    std::generate(h_filter, h_filter+OUTPUT_C*INPUT_C*FILTER_H*FILTER_W, [](){return (std::rand()%101-50)/10;});
    cudaErrChk( hipMemcpy(d_filter, h_filter, sizeof(float)*OUTPUT_C*INPUT_C*FILTER_H*FILTER_W, hipMemcpyHostToDevice) );
}




cuConvFloat::~cuConvFloat() {

    /******************************************************************
     * Finallize
     *******************************************************************/
    free (h_filter);

    cudaErrChk( hipFree(d_workspace_forward) );
    cudaErrChk( hipFree(d_workspace_backward_data) );
    cudaErrChk( hipFree(d_workspace_backward_filter) );
    
    cudaErrChk( hipFree(d_x) );
    cudaErrChk( hipFree(d_y) );
    cudaErrChk( hipFree(d_dx) );
    cudaErrChk( hipFree(d_dy) );
    cudaErrChk( hipFree(d_filter) );

    cudnnErrChk( hipdnnDestroyTensorDescriptor(desc_x) );
    cudnnErrChk( hipdnnDestroyTensorDescriptor(desc_y) );
    cudnnErrChk( hipdnnDestroyTensorDescriptor(desc_dx) );
    cudnnErrChk( hipdnnDestroyTensorDescriptor(desc_dy) );

    cudnnErrChk( hipdnnDestroyFilterDescriptor(desc_filter) );
    cudnnErrChk( hipdnnDestroyFilterDescriptor(desc_dw) );

}


void cuConvFloat::forward(float* x) {

    /******************************************************************
     * 6. Launch forward kernel
     *******************************************************************/
    const float alpha=1, beta=0;
    
    cudaErrChk( hipMemcpy(d_x, x, sizeof(float)*BATCH_NUM*INPUT_C*INPUT_H*INPUT_W, hipMemcpyDeviceToDevice) );

    cudnnErrChk( hipdnnConvolutionForward(*cudnn
                                        , /*ALPHA*/&alpha
                                        , /*INPUT*/desc_x, d_x
                                        , /*KERNEL*/desc_filter, d_filter
                                        , /*LAYER*/desc_conv2d, perf_conv2d_algo_forward.algo, d_workspace_forward, bytes_workspace_forward
                                        , /*BETA*/&beta
                                        , /*OUTPUT*/desc_y, d_y
                                    ) );
    cudaErrChk( hipDeviceSynchronize() );

}

void cuConvFloat::backward(float* dy) {

    /******************************************************************
     * 6. Launch backward kernel
     *******************************************************************/
    
    const float alpha=1, beta=0;
    cudnnErrChk( hipdnnConvolutionBackwardData(*cudnn
                                        , /*ALPHA*/&alpha
                                        , /*KERNEL*/desc_filter, d_filter
                                        , /*dy*/desc_dy, d_dy
                                        , /*LAYER*/desc_conv2d, perf_conv2d_algo_backward_data.algo, d_workspace_backward_data, bytes_workspace_backward_data
                                        , /*BETA*/&beta
                                        , /*dx*/desc_dx, d_dx
                                    ) );
    
    cudnnErrChk( hipdnnConvolutionBackwardFilter(*cudnn
                                        , /*ALPHA*/&alpha
                                        , /*x*/desc_x, d_x
                                        , /*dy*/desc_dy, d_dy
                                        , /*LAYER*/desc_conv2d, perf_conv2d_algo_backward_filter.algo, d_workspace_backward_filter, bytes_workspace_backward_filter
                                        , /*BETA*/&beta
                                        , /*dw*/desc_dw, d_dw
                                    ) );
    
    cudaErrChk( hipDeviceSynchronize() );
}

void cuConvFloat::set_weights(float* filter_) {
    memcpy(h_filter, filter_, sizeof(float)*OUTPUT_C*INPUT_C*FILTER_H*FILTER_W);
    cudaErrChk( hipMemcpy(d_filter, h_filter, sizeof(float)*OUTPUT_C*INPUT_C*FILTER_H*FILTER_W, hipMemcpyHostToDevice) );
}
